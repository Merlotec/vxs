#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include <hip/hip_runtime.h>
#include <iostream>

int init_cuda_thread() {
  hipInit(0);
  hipDevice_t dev;
  hipDeviceGet(&dev, 0);
  hipCtx_t ctx;
  hipCtxCreate(&ctx, 0, dev);
  return 0;
}

int test_vk_texture(int memFD, int semFD, int width, int height) {
  hipError_t err;

  int size = width * height * 4 * 4;

  // 1) Import the Vulkan memory FD into CUDA
  hipExternalMemory_t extMem = nullptr;
  hipExternalMemoryHandleDesc memDesc{};
  memDesc.type       = hipExternalMemoryHandleTypeOpaqueFd;
  memDesc.handle.fd  = memFD;
  memDesc.size       = size;
  memDesc.flags      = 0;
  err = hipImportExternalMemory(&extMem, &memDesc);
  if (err != hipSuccess) {
    std::cerr << "hipImportExternalMemory failed: " << err << "\n";
    return -1;
  }

  // 2) Describe & map level-0 of the image as a 2D int4 array
  hipMipmappedArray_t mip = nullptr;
  CUDA_EXTERNAL_MEMORY_MIPMAPPED_ARRAY_DESC arrDesc{};
  arrDesc.numLevels               = 1;
  arrDesc.offset = 0;
  arrDesc.arrayDesc.Width         = width;
  arrDesc.arrayDesc.Height        = height;
  arrDesc.arrayDesc.Depth         = 0;                       // 2D
  arrDesc.arrayDesc.Format        = HIP_AD_FORMAT_SIGNED_INT32;
  arrDesc.arrayDesc.NumChannels   = 4;                       // vec4<i32>
  arrDesc.arrayDesc.Flags         = CUDA_ARRAY3D_COLOR_ATTACHMENT;
  err = cuExternalMemoryGetMappedMipmappedArray(&mip, extMem, &arrDesc);
  if (err != hipSuccess) {
    const char* name   = nullptr;
    const char* desc   = nullptr;
    hipDrvGetErrorName(err,  &name);
    hipDrvGetErrorString(err, &desc);
    std::cerr << "cuExternalMemoryGetMappedMipmappedArray failed: " << err << name << desc << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // 3) Pull out the single level
  hipArray_t cuArray = nullptr;
  err = hipMipmappedArrayGetLevel(&cuArray, mip, 0);
  if (err != hipSuccess) {
    std::cerr << "hipMipmappedArrayGetLevel failed: " << err << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // 4) Import & wait on the Vulkan-signaled semaphore
  hipExternalSemaphore_t extSem = nullptr;
  hipExternalSemaphoreHandleDesc semDesc{};
  semDesc.type      = hipExternalSemaphoreHandleTypeOpaqueFd;
  semDesc.handle.fd = semFD;
  err = hipImportExternalSemaphore(&extSem, &semDesc);
  if (err != hipSuccess) {
    std::cerr << "hipImportExternalSemaphore failed: " << err << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  hipExternalSemaphoreWaitParams waitParams{};
  waitParams.flags = 0;
  // For a binary semaphore, fenceValue and other fields are ignored
  err = hipWaitExternalSemaphoresAsync(&extSem, &waitParams, 1, 0 /* default stream */);
  if (err != hipSuccess) {
    std::cerr << "hipWaitExternalSemaphoresAsync failed: " << err << "\n";
    hipDestroyExternalSemaphore(extSem);
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // Optionally sync the stream to be 100% sure before host reading
  hipStreamSynchronize(0);

  // … now you can use `cuArray` in kernels or with hipTexObjectCreate …

  // 5) Cleanup
  hipDestroyExternalSemaphore(extSem);
  hipDestroyExternalMemory(extMem);
  return 0;
}
