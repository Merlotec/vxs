#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include <hip/hip_runtime.h>
#include <iostream>


#define CHECK(call) do {                                 \
    hipError_t _e = (call);                                \
    if (_e != hipSuccess) {                            \
        const char *n = 0, *d = 0;                       \
        hipDrvGetErrorName(_e, &n);                          \
        hipDrvGetErrorString(_e, &d);                        \
        fprintf(stderr, "%s failed: %d %s - %s\n",       \
                #call, _e, n?n:"", d?d:"");              \
        return -1;                                       \
    }                                                    \
} while (0)

int init_cuda_thread() {
  hipInit(0);
  hipDevice_t dev;
  hipDeviceGet(&dev, 0);
  hipCtx_t ctx;
  hipCtxCreate(&ctx, 0, dev);
  return 0;
}

int test_vk_texture(int memFD, int semFD, int width, int height, uint64_t vk_allocation_size, uint64_t vk_bind_offset, int is_dedicated_allocation) {
  hipError_t err;

  // 1) Import the Vulkan memory FD into CUDA
  hipExternalMemory_t extMem = nullptr;
  hipExternalMemoryHandleDesc memDesc{};
  memDesc.type       = hipExternalMemoryHandleTypeOpaqueFd;
  memDesc.handle.fd  = memFD;
  memDesc.size       = vk_allocation_size;
  memDesc.flags      = is_dedicated_allocation ? hipExternalMemoryDedicated : 0;
  err = hipImportExternalMemory(&extMem, &memDesc);
  if (err != hipSuccess) {
    std::cerr << "hipImportExternalMemory failed: " << err << "\n";
    return -1;
  }

  // 2) Describe & map level-0 of the image as a 2D int4 array
  hipMipmappedArray_t mip = nullptr;
  CUDA_EXTERNAL_MEMORY_MIPMAPPED_ARRAY_DESC arrDesc{};
  arrDesc.numLevels               = 1;
  arrDesc.offset = vk_allocation_size;
  arrDesc.arrayDesc.Width         = width;
  arrDesc.arrayDesc.Height        = height;
  arrDesc.arrayDesc.Depth         = 0;                       // 2D
  arrDesc.arrayDesc.Format        = HIP_AD_FORMAT_SIGNED_INT32;
  arrDesc.arrayDesc.NumChannels   = 4;                       // vec4<i32>
  arrDesc.arrayDesc.Flags         |= CUDA_ARRAY3D_COLOR_ATTACHMENT;
  err = cuExternalMemoryGetMappedMipmappedArray(&mip, extMem, &arrDesc);
  if (err != hipSuccess) {
    const char* name   = nullptr;
    const char* desc   = nullptr;
    hipDrvGetErrorName(err,  &name);
    hipDrvGetErrorString(err, &desc);
    std::cerr << "cuExternalMemoryGetMappedMipmappedArray failed: " << err << name << desc << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // 3) Pull out the single level
  hipArray_t cuArray = nullptr;
  err = hipMipmappedArrayGetLevel(&cuArray, mip, 0);
  if (err != hipSuccess) {
    std::cerr << "hipMipmappedArrayGetLevel failed: " << err << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // 4) Import & wait on the Vulkan-signaled semaphore
  hipExternalSemaphore_t extSem = nullptr;
  hipExternalSemaphoreHandleDesc semDesc{};
  semDesc.type      = hipExternalSemaphoreHandleTypeOpaqueFd;
  semDesc.handle.fd = semFD;
  err = hipImportExternalSemaphore(&extSem, &semDesc);
  if (err != hipSuccess) {
    std::cerr << "hipImportExternalSemaphore failed: " << err << "\n";
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  hipExternalSemaphoreWaitParams waitParams{};
  waitParams.flags = 0;
  // For a binary semaphore, fenceValue and other fields are ignored
  err = hipWaitExternalSemaphoresAsync(&extSem, &waitParams, 1, 0 /* default stream */);
  if (err != hipSuccess) {
    std::cerr << "hipWaitExternalSemaphoresAsync failed: " << err << "\n";
    hipDestroyExternalSemaphore(extSem);
    hipDestroyExternalMemory(extMem);
    return -1;
  }

  // Optionally sync the stream to be 100% sure before host reading
  hipStreamSynchronize(0);

  // … now you can use `cuArray` in kernels or with hipTexObjectCreate …

  // 5) Cleanup
  hipDestroyExternalSemaphore(extSem);
  hipDestroyExternalMemory(extMem);
  return 0;
}

// Returns 0 on success; prints diagnostics on failures.
// Use exactly the Vulkan allocation size and bind offset you used on the Vulkan side.
int probe_cuda_import(
    int mem_fd,
    unsigned long long vk_allocation_size,
    unsigned long long vk_bind_offset,   // 0 if dedicated allocation
    unsigned width, unsigned height,
    int color_attachment,                // 1 if image used as color target
    int dedicated_allocation,            // 1 if VkMemoryDedicatedAllocateInfo was used
    int cu_fmt, unsigned num_channels  // e.g., HIP_AD_FORMAT_SIGNED_INT32, 4
) {
    // CHECK(hipInit(0));
    // hipDevice_t dev; CHECK(hipDeviceGet(&dev, 0));
    // hipCtx_t ctx; CHECK(hipCtxCreate(&ctx, 0, dev));

    // 1) Import external memory
    hipExternalMemory_t extMem = 0;
    hipExternalMemoryHandleDesc memDesc = {};
    memDesc.type      = hipExternalMemoryHandleTypeOpaqueFd;
    memDesc.handle.fd = mem_fd;                    // CUDA takes ownership on success
    memDesc.size      = vk_allocation_size;        // <-- EXACT Vulkan allocation size
    memDesc.flags     = dedicated_allocation ? hipExternalMemoryDedicated : 0;
    CHECK(hipImportExternalMemory(&extMem, &memDesc));

    // 2) BUFFER probe (sanity check for FD/size/offset)
    hipDeviceptr_t devPtr = 0;
    hipExternalMemoryBufferDesc b = {};
    b.offset = vk_bind_offset; // 0 for dedicated
    // map something modest, but aligned by construction (using the full tail is simplest)
    unsigned long long map_size = vk_allocation_size - vk_bind_offset;
    if (map_size > (1ull<<20)) map_size = (1ull<<20); // map up to 1 MiB for the probe
    b.size   = map_size;
    b.flags  = 0; // must be zero
    CHECK(hipExternalMemoryGetMappedBuffer(&devPtr, extMem, &b));  // may fail if size/offset wrong

    // Try reading a few bytes to host (proves the mapping is valid)
    unsigned char tmp[64];
    memset(tmp, 0xCD, sizeof(tmp));
    CHECK(hipMemcpyDtoH(tmp, devPtr, sizeof(tmp)));               // a small read should succeed

    // Free the temporary buffer mapping (required by CUDA; otherwise Destroy will complain)
    CHECK(hipFree(devPtr));  // driver API requires hipFree on mapped buffers
    // (Doc: buffers mapped from external memory must be freed with hipFree)  [oai_citation:2‡NVIDIA Docs](https://docs.nvidia.com/cuda/archive/11.4.4/pdf/CUDA_Driver_API.pdf)

    // 3) ARRAY probe (format/flags/extent correctness)
    hipMipmappedArray_t mip = 0;
    CUDA_EXTERNAL_MEMORY_MIPMAPPED_ARRAY_DESC mm = {};
    mm.offset    = vk_bind_offset;   // 0 if dedicated
    mm.numLevels = 1;

    HIP_ARRAY3D_DESCRIPTOR arr = {};
    arr.Width       = width;
    arr.Height      = height;
    arr.Depth       = 0;                       // 2D
    arr.Format      = (hipArray_Format)cu_fmt;                  // e.g., HIP_AD_FORMAT_SIGNED_INT32
    arr.NumChannels = num_channels;            // e.g., 4
    arr.Flags       = 0;
    if (color_attachment)
        arr.Flags |= CUDA_ARRAY3D_COLOR_ATTACHMENT; // REQUIRED for render targets  [oai_citation:3‡NVIDIA Docs](https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EXTRES__INTEROP.html)
    // Optional: if you’ll use CUDA surface writes, also add hipArraySurfaceLoadStore.

    mm.arrayDesc = arr;

    hipError_t r = cuExternalMemoryGetMappedMipmappedArray(&mip, extMem, &mm);
    if (r != hipSuccess) {
        const char *n=0,*d=0; hipDrvGetErrorName(r,&n); hipDrvGetErrorString(r,&d);
        fprintf(stderr, "cuExternalMemoryGetMappedMipmappedArray failed: %s - %s\n",
                n?n:"", d?d:"");
        // If buffer probe succeeded but array probe failed, it’s almost certainly format/flags.
        hipDestroyExternalMemory(extMem);
        // hipCtxDestroy(ctx);
        return -2;
    }

    // Pull level-0 and sanity-copy a tiny tile to host
    hipArray_t level0 = 0; CHECK(hipMipmappedArrayGetLevel(&level0, mip, 0));

    // Query back CUDA’s view of the array (helps confirm your descriptor matched)
    HIP_ARRAY_DESCRIPTOR arr_out = {};
    CHECK(hipArrayGetDescriptor(&arr_out, level0));
    fprintf(stderr, "CUDA array: %ux%u fmt=%u numCh=%u\n",
            (unsigned)arr_out.Width, (unsigned)arr_out.Height,
            (unsigned)arr_out.Format, (unsigned)arr_out.NumChannels);

    // Try a 16x16 read to host using hipMemcpyParam2D
    const unsigned tileW = (width  < 16 ? width  : 16);
    const unsigned tileH = (height < 16 ? height : 16);
    const size_t   elemB = 4 /*int32*/ * num_channels;  // vec4<i32> => 16 bytes/texel
    void* host = malloc(tileW * tileH * elemB);

    hip_Memcpy2D c2d = {};
    c2d.srcMemoryType = hipMemoryTypeArray;
    c2d.srcArray      = level0;
    c2d.dstMemoryType = hipMemoryTypeHost;
    c2d.dstHost       = host;
    c2d.dstPitch      = tileW * elemB;
    c2d.WidthInBytes  = tileW * elemB;
    c2d.Height        = tileH;
    CHECK(hipMemcpyParam2D(&c2d)); // if this works, the array is mapped and accessible

    // cleanup
    free(host);
    // No explicit hipArrayDestroy for external arrays; destroying the external memory drops it:
    CHECK(hipDestroyExternalMemory(extMem));
    // CHECK(hipCtxDestroy(ctx));
    return 0;
}
